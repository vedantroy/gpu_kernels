#include "hip/hip_runtime.h"
#include <iostream>
#include <mpi.h>
#include <hip/hip_runtime.h>

__global__ void writeData(int* data, int rank, int world_size) {
    if (threadIdx.x < world_size) {
        data[rank] = rank;  // Write rank to its own position
        printf("Rank %d writing to position %d\n", rank, rank);
    }
}

int main(int argc, char** argv) {
    MPI_Init(&argc, &argv);

    int world_size, rank;
    MPI_Comm_size(MPI_COMM_WORLD, &world_size);
    MPI_Comm_rank(MPI_COMM_WORLD, &rank);

    // Initialize CUDA
    hipSetDevice(rank);

    // Allocate memory for all ranks on each device
    int* data;
    hipMalloc((void**)&data, world_size * sizeof(int));
    hipIpcMemHandle_t handle;
    hipIpcGetMemHandle(&handle, data);

    // Gather all handles
    hipIpcMemHandle_t* handles = new hipIpcMemHandle_t[world_size];
    MPI_Allgather(&handle, sizeof(hipIpcMemHandle_t), MPI_BYTE, handles, sizeof(hipIpcMemHandle_t), MPI_BYTE, MPI_COMM_WORLD);

    // Write rank to each device's memory
    for (int i = 0; i < world_size; i++) {
        if (i != rank) { // Skip own memory
            int* remoteData;
            hipIpcOpenMemHandle((void**)&remoteData, handles[i], hipIpcMemLazyEnablePeerAccess);
            writeData<<<1, world_size>>>(remoteData, rank, world_size);
            hipDeviceSynchronize();
            hipIpcCloseMemHandle(remoteData);
        }
    }

    // Cleanup
    hipFree(data);
    delete[] handles;

    MPI_Finalize();
    return 0;
}