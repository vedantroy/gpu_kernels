#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>

__global__ void writeData(int* data, int rank, int world_size) {
    if (threadIdx.x < world_size) {
        data[rank] = rank;  // Write rank to its own position
        printf("Rank %d writing to position %d\n", rank, rank);
    }
}

extern "C" void gpu_work(int rank, int world_size) {
    // Initialize CUDA
    hipSetDevice(rank);

    // Allocate memory and create IPC handle
    int* data;
    hipMalloc((void**)&data, world_size * sizeof(int));
    hipIpcMemHandle_t handle;
    hipIpcGetMemHandle(&handle, data);

    // Gather all handles
    hipIpcMemHandle_t* handles = new hipIpcMemHandle_t[world_size];
    MPI_Allgather(&handle, sizeof(hipIpcMemHandle_t), MPI_BYTE, handles, sizeof(hipIpcMemHandle_t), MPI_BYTE, MPI_COMM_WORLD);

    // Call the kernel for each device's memory
    for (int i = 0; i < world_size; i++) {
        if (i != rank) {
            int* remoteData;
            hipIpcOpenMemHandle((void**)&remoteData, handles[i], hipIpcMemLazyEnablePeerAccess);
            writeData<<<1, world_size>>>(remoteData, rank, world_size);
            hipIpcCloseMemHandle(remoteData);
        }
    }

    // Cleanup
    hipFree(data);
    delete[] handles;

    hipDeviceSynchronize();
}